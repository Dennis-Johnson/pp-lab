#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAX_LEN 256

/* Lab 7: Programs on strings
*  Q1: Write a CUDA program to count the number of times
*     a given word is repeated in a sentence. (Use atomic function)
*/

__constant__ int len_sentence;
__constant__ int len_word;

__global__ void countWord(char *sentence, char *word, int *occurences){
    int indx = threadIdx.x;

    int flag = 0;
    for(int i = 0; i < len_word; i++){
        if(word[i] != sentence[i + indx]){
            flag = 1;
            break;
        }
    }

    if(flag == 0)
      atomicAdd(occurences, 1);
}

int main() {

  // host copies of sentence and word strings
  char sentence[MAX_LEN] = "This is a input sentence string string input string";
  char word[MAX_LEN] = "string";
  int len_sentence_h = strlen(sentence);
  int len_word_h = strlen(word);

  // number of occurences of 'word' in 'string'
  int occurences = 0;
 
  // device copies of variables
  char *d_sentence, *d_word;
  int  *d_occurences;

  size_t sizeSentence = len_sentence_h * sizeof(char);
  size_t sizeWord = len_word_h * sizeof(char);

  // Allocate space for device copies of variables
  hipMalloc((void **)&d_sentence, sizeSentence);
  hipMalloc((void **)&d_word, sizeWord);
  hipMalloc((void **)&d_occurences, sizeof(int));

  // Copy string lengths to constant memory on device
  hipMemcpyToSymbol(HIP_SYMBOL(len_sentence), &len_sentence_h, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(len_word), &len_word_h, sizeof(int));

  // Copy inputs to device
  hipMemcpy(d_sentence, sentence, sizeSentence, hipMemcpyHostToDevice);
  hipMemcpy(d_word, word, sizeWord, hipMemcpyHostToDevice);
  hipMemcpy(d_occurences, &occurences, sizeof(int), hipMemcpyHostToDevice);

  // Launch kernels on GPU:
  hipError_t err;

  // A thread for each possible starting index of 'word'
  int num_threads = len_sentence_h - len_word_h + 1;
  countWord<<<1, num_threads>>>(d_sentence, d_word, d_occurences);

  // Retrieve result
  err = hipMemcpy(&occurences, d_occurences, sizeof(int), hipMemcpyDeviceToHost);

  if(err != hipSuccess) 
    printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
  
  printf("Found %s in sentence %d times\n", word, occurences);

  // Free resources
  hipFree(d_sentence);
  hipFree(d_word);
  hipFree(d_occurences);

  return 0;
}
