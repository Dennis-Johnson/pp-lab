#include "hip/hip_runtime.h"
%%cu 
#include <stdio.h>
#include <hip/hip_runtime.h>

/* Lab8 Q2
* Write a program in CUDA to perform parallel Sparse Matrix Vector Multiplication 
* using compressed sparse row (CSR) storage format. 
* Represent the input sparse matrix in CSR format in the host code. */
__global__ void sparseMatMulCSR(
    int num_rows, 
    int *data, 
    int *col_index, 
    int *row_ptr,
    int *x,
    int *y
){  
  int row = threadIdx.x;
  if(row < num_rows){
      int accumulator = 0;
      int row_start = row_ptr[row];
      int row_end = row_ptr[row + 1];

      for(int i = row_start; i < row_end; i++)
        accumulator += data[i] * x[col_index[i]];

      y[row] = accumulator;
  }
}

int main(){
    const int n = 4;
    int y[n], row_ptr[n+1];
    int ipmat[n][n] = {{3,0,1,0},{0,0,0,0},{0,2,4,1},{1,0,0,1}};
    int x[] = {3,4,5,6};
    int nonzerocount = 0;
    /* Find the num of nonzero elements in input matrix
    *  Populate the row_ptr array
    */
    printf("Input matrix is \n");
    for(int i = 0; i < n; i++){
        row_ptr[i] = nonzerocount;
        for(int j = 0; j < n; j ++){
            if(ipmat[i][j] != 0)
              nonzerocount++;

            printf("%d ", ipmat[i][j]);
        }
        printf("\n");
    }
    // Last entry is total num of nonzero elements
    row_ptr[n] = nonzerocount;

    int data[nonzerocount], col_index[nonzerocount];
    int k = 0;

    /* Populate the col_index array
    *  Populte the data array
    */
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            if(ipmat[i][j] != 0){
                data[k] = ipmat[i][j];
                col_index[k++] = j;
            }
        }
    }
    
    printf("\nData array\n");
    for(int i = 0; i < nonzerocount; i++)
      printf("%d ", data[i]);

    printf("\nCol_Index array\n");
    for(int i = 0; i < nonzerocount; i++)
      printf("%d ", col_index[i]);

    printf("\nRow_Ptr array\n");
    for(int i = 0; i <= n; i++)
      printf("%d ", row_ptr[i]);

    printf("\nInput vector X to multiply\n");
    for(int i = 0; i < n; i++)
      printf("%d ", x[i]);

    // Device copies of inputs
    int *d_data, *d_col_index, *d_row_ptr, *d_x, *d_y;

    // Allocate memory on device
    hipMalloc((void**) &d_data, nonzerocount * sizeof(int));
    hipMalloc((void**) &d_col_index, nonzerocount * sizeof(int));
    hipMalloc((void**) &d_row_ptr, (n + 1) * sizeof(int));
    hipMalloc((void**) &d_x, n * sizeof(int));
    hipMalloc((void**) &d_y, n * sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_data, data, nonzerocount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, col_index, nonzerocount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch Kernel
    sparseMatMulCSR<<<1, n>>>(n, d_data, d_col_index, d_row_ptr, d_x, d_y);

    // Copy outputs back from device
    hipError_t err = hipMemcpy(y, d_y, n * sizeof(int), hipMemcpyDeviceToHost);
    if(err != hipSuccess) 
      printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));

    printf("\nThe transformed input vector is: \n");
    for(int i = 0; i < n; i++)
      printf("%d ", y[i]);

    // Free and cleanup
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_row_ptr);
    hipFree(d_x);
    hipFree(d_y);
    
    return 0;
}
